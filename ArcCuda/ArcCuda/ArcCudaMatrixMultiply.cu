#include "hip/hip_runtime.h"
// Cuda
#include "hip/hip_runtime.h"
#include ""

// Std C++
#include <stdio.h>
#include <iostream>

// Arc Cuda
#include "ArcCudaMatrixMultiply.h"

__global__ void matrixMultiply(int** ppMatrix1, int** ppMatrix2, int **ppMatrix3, const int matrixSizeM, const int matrixSizeN, const int matrixSizeP)
{
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int Pvalue = 0;

    //int runningResult = 0;
    //for (int k = 0; k < Width; k++)
    //{
    //    float Mdelement = Md[ty * Width + k];
    //    float Ndelement = Nd[k * Width + tx];
    //    // Pd[ty*Width+tx] += Mdelement * Ndelement; � NO!
    //    runningResult += Mdelement * Ndelement;
    //}
    //
    //Pd[ty * Width + tx] = runningResult;

    //for (int rowIndex = 0; rowIndex < matrixSizeM; ++rowIndex)
    //{
    //    for (int columnIndex = 0; columnIndex < matrixSizeP; ++columnIndex)
    //    {
    //        _ppMatrix3[rowIndex][columnIndex] = dotProduct(_ppMatrix1, _ppMatrix2, rowIndex, columnIndex, _matrixSizeN);
    //    }
    //}
    //ppMatrix3[tx][ty] = Pvalue;
}

bool calcMatrixMultiply(int** ppMatrix1, int** ppMatrix2, int** ppMatrix3, const int matrixSizeM, const int matrixSizeN, const int matrixSizeP)
{
    int** ppCudaMatrix1;
    int** ppCudaMatrix2;
    int** ppCudaMatrix3;

    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) 
    {
        std::cout << "Could not set cuda device.\n";
        return false;
    }

    cudaStatus = hipMalloc((void**)&ppCudaMatrix1, size_t(matrixSizeM) * size_t(matrixSizeN) * sizeof(int));
    if (cudaStatus != hipSuccess)
    {
        std::cout << "Could not allocate the first Cuda Matrix.\n";
        return false;
    }

    cudaStatus = hipMalloc((void**)&ppCudaMatrix2, size_t(matrixSizeN) * size_t(matrixSizeP) * sizeof(int));
    if (cudaStatus != hipSuccess)
    {
        std::cout << "Could not allocate the second Cuda Matrix.\n";
        return false;
    }

    cudaStatus = hipMalloc((void**)&ppCudaMatrix3, size_t(matrixSizeM) * size_t(matrixSizeP) * sizeof(int));
    if (cudaStatus != hipSuccess)
    {
        std::cout << "Could not allocate the third Cuda Matrix.\n";
        return false;
    }

    cudaStatus = hipMemcpy(ppCudaMatrix1, ppMatrix1, size_t(matrixSizeM) * size_t(matrixSizeN) * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) 
    {
        std::cout << "Could not copy the memory from the host first matrix to the device first Matrix.\n";
        return false;
    }

    cudaStatus = hipMemcpy(ppCudaMatrix2, ppMatrix2, size_t(matrixSizeN) * size_t(matrixSizeP) * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        std::cout << "Could not copy the memory from the host second matrix to the device second Matrix.\n";
        return false;
    }

    dim3 blockSize(matrixSizeN, matrixSizeN);
    dim3 gridSize(1, 1);

    matrixMultiply<<<gridSize, blockSize>>>(ppMatrix1, ppMatrix2, ppMatrix3, matrixSizeM, matrixSizeN, matrixSizeP);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) 
    {
        std::cout << "Error processing Cuda matrix multiplication.\n";
        return false;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) 
    {
        std::cout << "Error processing synchronizing Cuda kernel threads.\n";
        return false;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(ppCudaMatrix3, ppMatrix3, size_t(matrixSizeM) * size_t(matrixSizeP) * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) 
    {
        std::cout << "Could not copy the memory from the device third matrix to the host third matrix.\n";
        return false;
    }

    hipFree(ppCudaMatrix1);
    hipFree(ppCudaMatrix2);
    hipFree(ppCudaMatrix3);

    return true;
}