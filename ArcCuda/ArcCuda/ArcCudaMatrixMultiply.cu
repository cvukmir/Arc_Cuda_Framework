#include "hip/hip_runtime.h"
// Cuda
#include "hip/hip_runtime.h"
#include ""

// Std C++
#include <stdio.h>
#include <iostream>

// Arc Cuda
#include "ArcCudaMatrixMultiply.h"

__global__ void matrixMultiply(float* pMatrix1, float* pMatrix2, float* pMatrix3, const int matrixSizeM, const int matrixSizeN, const int matrixSizeP)
{
    int   threadX = threadIdx.x;
    int   threadY = threadIdx.y;
    float Pvalue  = 0.0f;

    for (int k = 0; k < matrixSizeM; ++k)
    {
        float Mdelement = pMatrix1[threadY * matrixSizeM + k];
        float Ndelement = pMatrix2[k * matrixSizeM + threadX];
        Pvalue += Mdelement * Ndelement;
    }

    pMatrix3[threadY * matrixSizeM + threadX] = Pvalue;
}

bool calcMatrixMultiply(float* pMatrix1, float* pMatrix2, float* pMatrix3, const int matrixSizeM, const int matrixSizeN, const int matrixSizeP)
{
    float* pCudaMatrix1;
    float* pCudaMatrix2;
    float* pCudaMatrix3;

    hipError_t cudaStatus;

    // Set the device //

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) 
    {
        std::cout << "Could not set cuda device.\n";
        return false;
    }

    // Allocate the arrays //

    cudaStatus = hipMalloc((void**)&pCudaMatrix1, size_t(matrixSizeM) * size_t(matrixSizeN) * sizeof(float));
    if (cudaStatus != hipSuccess)
    {
        std::cout << "Could not allocate the first Cuda Matrix.\n";
        return false;
    }

    cudaStatus = hipMalloc((void**)&pCudaMatrix2, size_t(matrixSizeN) * size_t(matrixSizeP) * sizeof(float));
    if (cudaStatus != hipSuccess)
    {
        hipFree(pCudaMatrix1);
        std::cout << "Could not allocate the second Cuda Matrix.\n";
        return false;
    }

    cudaStatus = hipMalloc((void**)&pCudaMatrix3, size_t(matrixSizeM) * size_t(matrixSizeP) * sizeof(float));
    if (cudaStatus != hipSuccess)
    {
        hipFree(pCudaMatrix1);
        hipFree(pCudaMatrix2);
        std::cout << "Could not allocate the third Cuda Matrix.\n";
        return false;
    }

    // Copy the memory from CPU to GPU //

    cudaStatus = hipMemcpy(pCudaMatrix1, pMatrix1, size_t(matrixSizeM) * size_t(matrixSizeN) * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) 
    {
        hipFree(pCudaMatrix1);
        hipFree(pCudaMatrix2);
        hipFree(pCudaMatrix3);
        std::cout << "Could not copy the memory from the host first matrix to the device first Matrix.\n";
        return false;
    }

    cudaStatus = hipMemcpy(pCudaMatrix2, pMatrix2, size_t(matrixSizeN) * size_t(matrixSizeP) * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        hipFree(pCudaMatrix1);
        hipFree(pCudaMatrix2);
        hipFree(pCudaMatrix3);
        std::cout << "Could not copy the memory from the host second matrix to the device second Matrix.\n";
        return false;
    }

    // Perform the multiplication //

    dim3 blockSize(matrixSizeN, matrixSizeN);
    dim3 gridSize(1, 1);

    matrixMultiply<<<gridSize, blockSize>>>(pCudaMatrix1, pCudaMatrix2, pCudaMatrix3, matrixSizeM, matrixSizeN, matrixSizeP);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) 
    {
        hipFree(pCudaMatrix1);
        hipFree(pCudaMatrix2);
        hipFree(pCudaMatrix3);
        std::cout << "Error processing Cuda matrix multiplication.\n";
        return false;
    }

    // Synchronize threads //

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) 
    {
        hipFree(pCudaMatrix1);
        hipFree(pCudaMatrix2);
        hipFree(pCudaMatrix3);
        std::cout << "Error processing synchronizing Cuda kernel threads.\n";
        return false;
    }

    // Copy the memory from the GPU to the CPU //
    
    cudaStatus = hipMemcpy(pMatrix3, pCudaMatrix3, size_t(matrixSizeM) * size_t(matrixSizeP) * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) 
    {
        hipFree(pCudaMatrix1);
        hipFree(pCudaMatrix2);
        hipFree(pCudaMatrix3);
        std::cout << "Could not copy the memory from the device third matrix to the host third matrix.\n";
        return false;
    }

    hipFree(pCudaMatrix1);
    hipFree(pCudaMatrix2);
    hipFree(pCudaMatrix3);

    return true;
}