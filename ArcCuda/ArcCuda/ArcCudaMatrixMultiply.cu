#include "hip/hip_runtime.h"
// Cuda
#include "hip/hip_runtime.h"
#include ""

// Std C++
#include <stdio.h>
#include <iostream>
#include <math.h>
#include <stdint.h>


// Arc Cuda
#include "ArcCudaMatrixMultiply.h"

const int BLOCK_WIDTH = 32; // AKA: TILE_WIDTH

__global__ void matrixMultiplyDynamic(float* pMatrix1, float* pMatrix2, float* pMatrix3, const int matrixSizeM, const int matrixSizeN, const int matrixSizeP)
{
    int   blockX           = blockIdx.x;
    int   blockY           = blockIdx.y;
    int   threadX          = threadIdx.x;
    int   threadY          = threadIdx.y;
    float computedValue    = 0.0f;

    int row = blockY * BLOCK_WIDTH + threadY;
    int col = blockX * BLOCK_WIDTH + threadX;

    if (row >= matrixSizeM || col >= matrixSizeP)
    {
        return;
    }

    for (int k = 0; k < matrixSizeN; ++k)
    {
        computedValue += pMatrix1[row * matrixSizeN + k] * pMatrix2[k * matrixSizeP + col];
    }

    pMatrix3[row * matrixSizeP + col] = computedValue;
}

__global__ void matrixMultiplyTiledBlocksSameSize(float* pMatrix1, float* pMatrix2, float* pMatrix3, const int matrixSizeM, const int matrixSizeN, const int matrixSizeP)
{
    //int   threadsPerXBlock = blockDim.x;
    //int   threadsPerYBlock = blockDim.y;
    int   blockX           = blockIdx.x;
    int   blockY           = blockIdx.y;
    int   threadX          = threadIdx.x;
    int   threadY          = threadIdx.y;
    float computedValue    = 0.0f;

    int row = blockY * BLOCK_WIDTH + threadY;
    int col = blockX * BLOCK_WIDTH + threadX;

    for (int k = 0; k < matrixSizeM; ++k)
    {
        computedValue += pMatrix1[row * matrixSizeM + k] * pMatrix2[k * matrixSizeM + col];
    }

    pMatrix3[row * matrixSizeM + col] = computedValue;
}

__global__ void matrixMultiplySameSize(float* pMatrix1, float* pMatrix2, float* pMatrix3, const int matrixSizeM, const int matrixSizeN, const int matrixSizeP)
{
    int   threadX          = threadIdx.x;
    int   threadY          = threadIdx.y;
    float computedValue    = 0.0f;

    for (int k = 0; k < matrixSizeM; ++k)
    {
        float matrix1Element = pMatrix1[threadY * matrixSizeM + k];
        float matrix2Element = pMatrix2[k * matrixSizeM + threadX];
        computedValue += matrix1Element * matrix2Element;
    }

    pMatrix3[threadY * matrixSizeM + threadX] = computedValue;
}

bool calcMatrixMultiply(float* pMatrix1, float* pMatrix2, float* pMatrix3, const int matrixSizeM, const int matrixSizeN, const int matrixSizeP)
{
    float* pCudaMatrix1;
    float* pCudaMatrix2;
    float* pCudaMatrix3;

    hipError_t cudaStatus;

    // Set the device //

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) 
    {
        std::cout << "Could not set cuda device.\n";
        return false;
    }

    // Allocate the arrays //

    cudaStatus = hipMalloc((void**)&pCudaMatrix1, size_t(matrixSizeM) * size_t(matrixSizeN) * sizeof(float));
    if (cudaStatus != hipSuccess)
    {
        std::cout << "Could not allocate the first Cuda Matrix.\n";
        return false;
    }

    cudaStatus = hipMalloc((void**)&pCudaMatrix2, size_t(matrixSizeN) * size_t(matrixSizeP) * sizeof(float));
    if (cudaStatus != hipSuccess)
    {
        hipFree(pCudaMatrix1);
        std::cout << "Could not allocate the second Cuda Matrix.\n";
        return false;
    }

    cudaStatus = hipMalloc((void**)&pCudaMatrix3, size_t(matrixSizeM) * size_t(matrixSizeP) * sizeof(float));
    if (cudaStatus != hipSuccess)
    {
        hipFree(pCudaMatrix1);
        hipFree(pCudaMatrix2);
        std::cout << "Could not allocate the third Cuda Matrix.\n";
        return false;
    }

    // Copy the memory from CPU to GPU //

    cudaStatus = hipMemcpy(pCudaMatrix1, pMatrix1, size_t(matrixSizeM) * size_t(matrixSizeN) * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) 
    {
        hipFree(pCudaMatrix1);
        hipFree(pCudaMatrix2);
        hipFree(pCudaMatrix3);
        std::cout << "Could not copy the memory from the host first matrix to the device first Matrix.\n";
        return false;
    }

    cudaStatus = hipMemcpy(pCudaMatrix2, pMatrix2, size_t(matrixSizeN) * size_t(matrixSizeP) * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        hipFree(pCudaMatrix1);
        hipFree(pCudaMatrix2);
        hipFree(pCudaMatrix3);
        std::cout << "Could not copy the memory from the host second matrix to the device second Matrix.\n";
        return false;
    }

    // Perform the multiplication //
    
    // Same size
    //dim3 blockSize(matrixSizeN, matrixSizeN);
    //dim3 gridSize(1, 1);
    //
    //matrixMultiplySameSize << <gridSize, blockSize >> > (pCudaMatrix1, pCudaMatrix2, pCudaMatrix3, matrixSizeM, matrixSizeN, matrixSizeP);

    // Tiled blocks - same size
    //dim3 threadsPerBlock(BLOCK_WIDTH, BLOCK_WIDTH);
    //dim3 numBlocks(std::ceil(matrixSizeM / static_cast<float>(BLOCK_WIDTH)), std::ceil(matrixSizeM / static_cast<float>(BLOCK_WIDTH)));
    //
    //matrixMultiplyTiledBlocks<<<numBlocks, threadsPerBlock>>>(pCudaMatrix1, pCudaMatrix2, pCudaMatrix3, matrixSizeM, matrixSizeN, matrixSizeP);

    // Dynamic sizes
    // Potential way of solving variable size:
    // x = Number of cols + block_width � 1 / block_width
    // y = Number of rows + block_width � 1 / block_width
    // numRows = sqrt(x) 
    // numCols = sqrt(y) 
    // float decimal;
    // numRows = std::modf()
    // if num is not exactly even, round up
    // dim3 numBlocks (num, num)

    dim3 threadsPerBlock(BLOCK_WIDTH, BLOCK_WIDTH);
    dim3 numBlocks(ceil(matrixSizeP / float(BLOCK_WIDTH)), ceil(matrixSizeM / float(BLOCK_WIDTH)));

    matrixMultiplyDynamic<<<numBlocks, threadsPerBlock>>>(pCudaMatrix1, pCudaMatrix2, pCudaMatrix3, matrixSizeM, matrixSizeN, matrixSizeP);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) 
    {
        hipFree(pCudaMatrix1);
        hipFree(pCudaMatrix2);
        hipFree(pCudaMatrix3);
        std::cout << "Error processing Cuda matrix multiplication.\n";
        return false;
    }

    // Copy the memory from the GPU to the CPU //
    
    cudaStatus = hipMemcpy(pMatrix3, pCudaMatrix3, size_t(matrixSizeM) * size_t(matrixSizeP) * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) 
    {
        hipFree(pCudaMatrix1);
        hipFree(pCudaMatrix2);
        hipFree(pCudaMatrix3);
        std::cout << "Could not copy the memory from the device third matrix to the host third matrix.\n";
        return false;
    }

    hipFree(pCudaMatrix1);
    hipFree(pCudaMatrix2);
    hipFree(pCudaMatrix3);

    return true;
}

bool isEvenDecimal(float value, float precision)
{
    float remainder = 0.0;
    modf(value, &remainder);

    return fabs((value + remainder) - value) < precision;
}