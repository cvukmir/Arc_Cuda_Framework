#include "hip/hip_runtime.h"
// Cuda
#include "hip/hip_runtime.h"
#include ""

// Std C++
#include <stdio.h>
#include <iostream>
#include <math.h>
#include <stdint.h>
#include <chrono>

// ArcCuda
#include "ArcCudaNormalization.h"

const int BLOCK_WIDTH = 1024; // AKA: TILE_WIDTH

__global__ void normalizationKernel(float* pInputArray, int size, float* normalizedValue)
{
	__shared__ float partialSum[BLOCK_WIDTH];

	unsigned int threadX = threadIdx.x;
	unsigned int blockX  = blockIdx.x;

	for (int blockIndex = 0; blockIndex <= size / BLOCK_WIDTH; ++blockIndex)
	{
		partialSum[threadX + (blockX * BLOCK_WIDTH)] = pInputArray[threadX + (blockX * BLOCK_WIDTH)] * pInputArray[threadX + (blockX * BLOCK_WIDTH)];

		for (unsigned int stride = BLOCK_WIDTH >> 1; stride > 0; stride >>= 1)
		{
			__syncthreads();
			if (threadX < stride)
			{
				partialSum[threadX] += partialSum[threadX + stride];
			}
		}

		__syncthreads();
	}

	if (threadX + (blockX * BLOCK_WIDTH) >= size)
	{
		return;
	}

	(*normalizedValue) += partialSum[0];

	if (threadX != 0 && blockX != 0)
	{
		return;
	}

	(*normalizedValue) = std::sqrtf(*normalizedValue);
}


bool calcNormalization(float* pArray, const int size, float* normalizedValue)
{
	float* pCudaArray;
	float* cudaNormalizedValue;

	hipError_t cudaStatus;

	// Set the device //

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) 
	{
		std::cout << "Could not set Cuda device.\n";
		return false;
	}

	// Allocate the arrays //

	cudaStatus = hipMalloc((void**)&pCudaArray, size_t(size) * sizeof(float));
	if (cudaStatus != hipSuccess)
	{
		std::cout << "Could not allocate the first Cuda array.\n";
		return false;
	}

	cudaStatus = hipMalloc((void**)&cudaNormalizedValue, sizeof(float));
	if (cudaStatus != hipSuccess)
	{
		std::cout << "Could not allocate the first Cuda array.\n";
		return false;
	}

	// Copy the memory from CPU to GPU //

	cudaStatus = hipMemcpy(pCudaArray, pArray, size_t(size) * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) 
	{
		hipFree(pCudaArray);
		std::cout << "Could not copy the memory from the host first array to the device first array.\n";
		return false;
	}

	// Perform the normalization //

	dim3 threadsPerBlock(BLOCK_WIDTH);
	dim3 numBlocks(ceil(size / float(BLOCK_WIDTH)));
	
	std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();

	normalizationKernel<<<numBlocks, threadsPerBlock>>>(pCudaArray, size, cudaNormalizedValue);

	std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();

	std::cout << "Normalization - Generated From GPU in " << std::chrono::duration_cast<std::chrono::nanoseconds>(end - begin).count() << " nanoseconds." << '\n';

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) 
	{
		hipFree(pCudaArray);
		std::cout << "Error processing Cuda array normalization.\n";
		return false;
	}

	// Copy the memory from the GPU to the CPU //
	
	cudaStatus = hipMemcpy(normalizedValue, cudaNormalizedValue, sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) 
	{
		hipFree(pCudaArray);
		std::cout << "Could not copy the memory from the device third arra to the host third array.\n";
		return false;
	}

	hipFree(pCudaArray);

	return true;
}