#include "hip/hip_runtime.h"
// Cuda
#include "hip/hip_runtime.h"
#include ""

// Std C++
#include <stdio.h>
#include <iostream>
#include <math.h>
#include <stdint.h>
#include <chrono>

// ArcCuda
#include "ArcCudaNormalization.h"

const int BLOCK_WIDTH = 1024; // AKA: TILE_WIDTH

__global__ void normalizationKernel(float* pInputArray, int size, float* normalizedValue)
{
	__shared__ float partialSum[BLOCK_WIDTH];

	unsigned int threadX = threadIdx.x;
	unsigned int blockX  = blockIdx.x;

	for (int blockIndex = 0; blockIndex <= size / BLOCK_WIDTH; ++blockIndex)
	{
		if ((blockIndex * BLOCK_WIDTH) + threadX < size)
		{
			partialSum[threadX] = pInputArray[(blockIndex * BLOCK_WIDTH) + threadX] * pInputArray[(blockIndex * BLOCK_WIDTH) + threadX];
		}
		else
		{
			partialSum[threadX] = 0.0;
		}

		__syncthreads();

		for (unsigned int stride = BLOCK_WIDTH >> 1; stride > 0; stride >>= 1)
		{
			if (threadX < stride)
			{
				partialSum[threadX] += partialSum[threadX + stride];
			}

			__syncthreads();
		}

		if (threadX == 0)
		{
			(*normalizedValue) += partialSum[0];
		}

		__syncthreads();
	}

	if (threadX == 0)
	{
		(*normalizedValue) = std::sqrtf((*normalizedValue));
	}
}


bool calcNormalization(float* pArray, const int size, float* normalizedValue)
{
	float* pCudaArray;
	float* cudaNormalizedValue;

	hipError_t cudaStatus;

	// Set the device //

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) 
	{
		std::cout << "Could not set Cuda device.\n";
		return false;
	}

	// Allocate the arrays //

	cudaStatus = hipMalloc((void**)&pCudaArray, size_t(size) * sizeof(float));
	if (cudaStatus != hipSuccess)
	{
		std::cout << "Could not allocate the first Cuda array.\n";
		return false;
	}

	cudaStatus = hipMalloc((void**)&cudaNormalizedValue, sizeof(float));
	if (cudaStatus != hipSuccess)
	{
		std::cout << "Could not allocate the Cuda normalized value.\n";
		return false;
	}

	// Copy the memory from CPU to GPU //

	cudaStatus = hipMemcpy(pCudaArray, pArray, size_t(size) * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) 
	{
		hipFree(pCudaArray);
		std::cout << "Could not copy the memory from the host first array to the device first array.\n";
		return false;
	}

	// Perform the normalization //

	dim3 threadsPerBlock(BLOCK_WIDTH);
	dim3 numBlocks(1);//ceil(size / float(BLOCK_WIDTH)));
	
	std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();

	normalizationKernel<<<numBlocks, threadsPerBlock>>>(pCudaArray, size, cudaNormalizedValue);

	std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();

	std::cout << "Normalization - Generated From GPU in " << std::chrono::duration_cast<std::chrono::nanoseconds>(end - begin).count() << " nanoseconds." << '\n';

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) 
	{
		hipFree(pCudaArray);
		std::cout << "Error processing Cuda array normalization.\n";
		return false;
	}

	// Copy the memory from the GPU to the CPU //
	
	cudaStatus = hipMemcpy(normalizedValue, cudaNormalizedValue, sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) 
	{
		hipFree(pCudaArray);
		std::cout << "Could not copy the memory from the device normalized value to the host normalized value.\n";
		return false;
	}

	hipFree(pCudaArray);

	return true;
}