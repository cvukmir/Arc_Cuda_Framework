#include "hip/hip_runtime.h"
// Cuda
#include "hip/hip_runtime.h"
#include ""

// Std C++
#include <stdio.h>
#include <iostream>
#include <math.h>
#include <stdint.h>
#include <chrono>

// ArcCuda
#include "ArcCudaNormalization.h"

const int BLOCK_WIDTH = 32; // AKA: TILE_WIDTH

__global__ void normalizationKernel(float* pInputArray, int size, float* normalizedValue)
{
	(*normalizedValue) = 10.0;
}


bool calcNormalization(float* pArray, const int size, float* normalizedValue)
{
	float* pCudaArray;
	float* cudaNormalizedValue;

	hipError_t cudaStatus;

	// Set the device //

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) 
	{
		std::cout << "Could not set cuda device.\n";
		return false;
	}

	// Allocate the arrays //

	cudaStatus = hipMalloc((void**)&pCudaArray, size_t(size) * sizeof(float));
	if (cudaStatus != hipSuccess)
	{
		std::cout << "Could not allocate the first Cuda array.\n";
		return false;
	}

	cudaStatus = hipMalloc((void**)&cudaNormalizedValue, sizeof(float));
	if (cudaStatus != hipSuccess)
	{
		std::cout << "Could not allocate the first Cuda array.\n";
		return false;
	}

	// Copy the memory from CPU to GPU //

	cudaStatus = hipMemcpy(pCudaArray, pArray, size_t(size) * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) 
	{
		hipFree(pCudaArray);
		std::cout << "Could not copy the memory from the host first matrix to the device first array.\n";
		return false;
	}

	// Perform the normalization //

	dim3 threadsPerBlock(1, 1);
	dim3 numBlocks(1, 1);
	//dim3 numBlocks(ceil(matrixSizeP / float(BLOCK_WIDTH)), ceil(matrixSizeM / float(BLOCK_WIDTH)));

	std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();

	normalizationKernel<<<numBlocks, threadsPerBlock>>>(pCudaArray, size, cudaNormalizedValue);

	std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();

	std::cout << "Normalization - Generated From GPU in " << std::chrono::duration_cast<std::chrono::nanoseconds>(end - begin).count() << " nanoseconds." << '\n';

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) 
	{
		hipFree(pCudaArray);
		std::cout << "Error processing Cuda array normalization.\n";
		return false;
	}

	// Copy the memory from the GPU to the CPU //
	
	cudaStatus = hipMemcpy(normalizedValue, cudaNormalizedValue, sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) 
	{
		hipFree(pCudaArray);
		std::cout << "Could not copy the memory from the device third matrix to the host third matrix.\n";
		return false;
	}

	hipFree(pCudaArray);

	return true;
}